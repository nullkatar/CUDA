#include <stdlib.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <locale.h>


#define BLOCK_SIZE 250
#define GRID_SIZE 4
#define THREAD_SIZE 1000
#define CUDA_FLOAT float

__global__ void pi_kern(CUDA_FLOAT *res)
{
        int n = threadIdx.x + blockIdx.x * BLOCK_SIZE;
        CUDA_FLOAT x0 = n * 1.f / (BLOCK_SIZE * GRID_SIZE);
        CUDA_FLOAT y0 = sqrt(1 - x0 * x0);
        CUDA_FLOAT dx = 1.f / (1.f * BLOCK_SIZE * GRID_SIZE);
        CUDA_FLOAT s = 0;
        CUDA_FLOAT x1, y1;
//      for(int i = 0; i < THREAD_SIZE; i++){
                x1 = x0 + dx; 
                y1 = sqrt(1 - x1 * x1);
                s += (y0 + y1) * dx / 2.f;
                x0 = x1;
                y0 = y1;
//      }
        res[n] = s;
}


__global__ void pi_kern_(CUDA_FLOAT *res_)
{
        int n = threadIdx.x + blockIdx.x * BLOCK_SIZE;
        CUDA_FLOAT x0 = n * 1.f / (BLOCK_SIZE * GRID_SIZE);
        CUDA_FLOAT y0 = x0 * sqrt(1 - x0 * x0);
        CUDA_FLOAT dx = 1.f / (1.f * BLOCK_SIZE * GRID_SIZE);
        CUDA_FLOAT s = 0;
        CUDA_FLOAT x1, y1;

//      for(int i = 0; i < THREAD_SIZE; i++){
        x1 = x0 + dx;
        y1 = x1 * sqrt(1 - x1 * x1);
        s += (y0 + y1) * dx / 2.f;
        x0 = x1;
        y0 = y1;
//      }
        res_[n] = s;
}


int main() {
        float *res_h,  *res_d_;
        float *res_d;
        res_h = (float *)malloc(sizeof(float)*THREAD_SIZE);
        hipMalloc((void **) &res_d, sizeof(float)*THREAD_SIZE);
        hipMalloc((void **) &res_d_, sizeof(float)*THREAD_SIZE);
        hipMemcpy(res_d, res_h, sizeof(float)*THREAD_SIZE, hipMemcpyHostToDevice);
        dim3 grid(GRID_SIZE);
        dim3 block(BLOCK_SIZE);
        pi_kern<<< grid,  block >>>(res_d);
        hipMemcpy(res_h, res_d, sizeof(float)*THREAD_SIZE, hipMemcpyDeviceToHost);

        float sum = 0.0;
        for(int i = 0; i < THREAD_SIZE; i++){
                sum += res_h[i];
        }

        hipMemcpy(res_d_, res_h, sizeof(float)*THREAD_SIZE, hipMemcpyHostToDevice);
        pi_kern_<<< grid,  block >>>(res_d_);
        hipMemcpy(res_h, res_d_, sizeof(float)*THREAD_SIZE, hipMemcpyDeviceToHost);

        float sum_ = 0.0;
        for(int j = 0; j < THREAD_SIZE; j++){
                sum_ += res_h[j];
        }

        printf("%0.8f\n", sum*4);
        printf("%0.8f\n", sum_ / sum);

        free(res_h);
        hipFree(res_d);
        hipFree(res_d_);
        return 0;

}

